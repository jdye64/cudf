/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <kernel.cu>
#include <kernel_wrapper.hpp>
#include <assert.h>
#include <cstdio>
#include <cudf/column/column_device_view.cuh>


CudfWrapper::CudfWrapper(cudf::mutable_table_view table_view) {
  mtv = table_view;
}

void CudfWrapper::tenth_mm_to_inches(int column_index) {

  // Example of showing num_columns and num_rows only for potential debugging
  printf("kernel_wrapper.cu # of columns: %lu\n", mtv.num_columns());
  printf("kernel_wrapper.cu # of rows: %lu\n", mtv.num_rows());

  //print out column dtypes for example sake only, not required,
  std::for_each( mtv.cbegin(), mtv.cend(), [](auto c) {
    printf("%d type=%d, ptr=%p\n", c, static_cast<int>(c.type().id()), c.data<char>() );
  });

  std::unique_ptr<cudf::mutable_column_device_view, std::function<void(cudf::mutable_column_device_view*)>> 
  mutable_device_column = cudf::mutable_column_device_view::create(mtv.column(column_index));

  // Invoke the Kernel to convert tenth_mm -> inches
  kernel_tenth_mm_to_inches<<<(mtv.num_rows()+255)/256, 256>>>(*mutable_device_column);
  hipDeviceSynchronize();
}

CudfWrapper::~CudfWrapper() {
  // It is important to note that CudfWrapper does not own the underlying Dataframe 
  // object and that will be freed by the Python/Cython layer later.
}
